

#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void add(char *A,int len) {
		int idx=threadIdx.x;
		// int ctr=0;
		int rel_size=len;

		if(len%2!=0)
			rel_size++;


		if(len%2==0){	
			char temp=A[idx];
			A[idx]=A[rel_size -(idx+1)];
			A[rel_size -(idx+1)]=temp;
		}
		else{
			char temp=A[idx];
			A[idx]=A[rel_size -(idx+1)-1];
			A[rel_size -(idx+1)-1]=temp;
			
		}
		
}
int main(void) {
	char c[100];
	char sub[100];
	int *o;
	// host copies of variables a, b & c
	char *A;
	char *d_sub;
	int *d_o;
	int d_sub_len; // device copies of variables a, b & c
	// int size = sizeof(int);
	printf("Enter String\n");
	fgets (c, 100, stdin);
		printf("%s\n",c);
	int size=strlen(c)-1;
	// printf("MAINS TIRGN %d\n",size );

	// printf("Enter Sub String\n");
	// fgets (sub, 100, stdin);
		printf("%s\n",sub);
	// int size_sub=strlen(sub)-1;
	// d_sub_len=size_sub;

	// int size_o=size-size_sub+1;
	// Allocate space for device copies of a, b, c
	hipMalloc((void **)&A, size*sizeof(char));
	// cudaMalloc((void **)&d_sub, size_sub*sizeof(char));
	// cudaMalloc((void **)&d_o, size_o*sizeof(int));
	// o=(int*)calloc(size_o,sizeof(int));
	// o={0};
	// cudaMalloc((void **)&o, sizeof(int));
	// cudaMalloc((void **)d_sub_len, sizeof(int));
	
	// Setup input values
	// Copy inputs to device
	hipMemcpy(A, c, size*sizeof(char), hipMemcpyHostToDevice);

	// cudaMemcpy(d_sub, sub, size_sub*sizeof(char), cudaMemcpyHostToDevice);
	// cudaMemcpy(d_b, &b, size, cudaMemcpyHostToDevice);
	// Launch add() kernel on GPU
	int rel_size=size;
	if(size%2!=0)
		rel_size++;
	add<<<1,rel_size>>>(A, size);
	// Copy result back to host
	hipMemcpy(c, A, size*sizeof(char), hipMemcpyDeviceToHost);
		
	printf("%s\n",c );
	int temp=0;
	// printf("LOLOL %d\n",size_o);
	/*for (int i = 0; i < size_o; ++i)
	{
		if(o[i]!=0){
			temp=i;
			break;
		}
		else
			temp=0;
	}
	
	printf("LOLOL\n");
	
	if(temp!=0){
		int pos;
		if(temp==-1)
			pos=0;
		else
			pos=temp;
		printf("found at %d\n",pos);
	}else{
		printf("Not found\n");
	}*/

	// printf("%d\n",c );
	// Cleanup
	// cudaFree(d_o);

	hipFree(A);
	// cudaFree(d_c);
	return 0;
}