

#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void add(char *c, char *sub, int *o,int sub_len) {
		int idx=threadIdx.x;
		int ctr=0;

		for (int i = 0; i < sub_len; ++i)
		{
			if(c[idx+i]==sub[i])
				ctr++;

		}
		o[idx]=0;
		if(idx==0 && ctr==sub_len)
			o[idx]=-1;
		else if(ctr==sub_len)
			o[idx]=1;
		
}
int main(void) {
	char c[100];
	char sub[100];
	int *o;
	// host copies of variables a, b & c
	char *d_c;
	char *d_sub;
	int *d_o;
	int d_sub_len; // device copies of variables a, b & c
	// int size = sizeof(int);
	printf("Enter String\n");
	fgets (c, 100, stdin);
		printf("%s\n",c);
	int size=strlen(c)-1;
	// printf("MAINS TIRGN %d\n",size );

	printf("Enter Sub String\n");
	fgets (sub, 100, stdin);
		printf("%s\n",sub);
	int size_sub=strlen(sub)-1;
	d_sub_len=size_sub;

	int size_o=size-size_sub+1;
	// Allocate space for device copies of a, b, c
	hipMalloc((void **)&d_c, size*sizeof(char));
	hipMalloc((void **)&d_sub, size_sub*sizeof(char));
	hipMalloc((void **)&d_o, size_o*sizeof(int));
	o=(int*)calloc(size_o,sizeof(int));
	// o={0};
	// cudaMalloc((void **)&o, sizeof(int));
	// cudaMalloc((void **)d_sub_len, sizeof(int));
	
	// Setup input values
	// Copy inputs to device
	hipMemcpy(d_c, c, size*sizeof(char), hipMemcpyHostToDevice);

	hipMemcpy(d_sub, sub, size_sub*sizeof(char), hipMemcpyHostToDevice);
	// cudaMemcpy(d_b, &b, size, cudaMemcpyHostToDevice);
	// Launch add() kernel on GPU

	add<<<1,size_o>>>(d_c, d_sub, d_o,d_sub_len);
	// Copy result back to host
	hipMemcpy(o, d_o, size_o*sizeof(int), hipMemcpyDeviceToHost);
	
	int temp=0;
	// printf("LOLOL %d\n",size_o);
	for (int i = 0; i < size_o; ++i)
	{
		if(o[i]!=0){
			temp++;
			// break;
		}
	}
	
	printf("LOLOL\n");
	
	if(temp!=0){
		int pos;
		if(temp==-1)
			pos=0;
		else
			pos=temp;
		printf("No of times found %d\n",pos);
	}else{
		printf("Not found\n");
	}

	// printf("%d\n",c );
	// Cleanup
	hipFree(d_o);

	hipFree(d_sub);
	hipFree(d_c);
	return 0;
}