
#include <hip/hip_runtime.h>
#include "malloc.h"
#include <stdio.h>

__global__ void add(int *a, int *b, int *c,int *col) {
	// *c = *a + *b
	int id=blockIdx.x;
	int start=id*col[0];
	for(int i=0;i<col[0];i++){
		c[start+i]=a[start+i]+b[start+i];
	}

}
int main(void) {
	int *a, *b, *c;
	int numR,numC;
	printf("Enter number of rows and columns :\n");
	scanf("%d",&numR);
	scanf("%d",&numC);
	int size = sizeof(int);

	// a=(int*)malloc(numC*numR*size);
	// b=(int*)malloc(numC*numR*size);
	// c=(int*)malloc(numC*numR*size);
	hipMalloc((void **)&b, numC*numR*size);
	hipMalloc((void **)&a, numC*numR*size);
	hipMalloc((void **)&c, numC*numR*size);


	printf("Enter A\n");
	int temp;
	for (int i = 0; i < numC*numR; ++i)
	{	
		scanf("%d",&temp);
		a[i]=temp;
	}
	printf("Enter B\n");
	for (int i = 0; i < numC*numR; ++i)
	{	
		scanf("%d",&temp);
		b[i]=temp;
	}	
	// host copies of variables a, b & c
	int *d_a, *d_b, *d_c,*col; // device copies of variables a, b & c
	// Allocate space for device copies of a, b, c
	hipMalloc((void **)&d_a, numC*numR*size);
	hipMalloc((void **)&d_b, numC*numR*size);
	hipMalloc((void **)&d_c, numC*numR*size);
	hipMalloc((void **)&col, size);
	
	// Setup input values
	
	// Copy inputs to device
	hipMemcpy(d_a, &a, numC*numR*size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, numC*numR*size, hipMemcpyHostToDevice);
	hipMemcpy(col,&numC,size,hipMemcpyHostToDevice);
	// Launch add() kernel on GPU
	printf("LOOOL\n");
	add<<<numR,1>>>(d_a, d_b, d_c,col);
	// Copy result back to host
	hipMemcpy(&c, d_c,numC*numR*size, hipMemcpyDeviceToHost);
	printf("LOOOL\n");
	
	for (int i = 0; i < numC*numR; ++i)
	{	
		if(i%numC==0)
			printf("wot\n");
		printf("%d ",c[i]);
	}	
	printf("LOOOL\n");
	// printf("ANS	%d\t%d\n",c[0],c[1] );

	// Cleanup
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}