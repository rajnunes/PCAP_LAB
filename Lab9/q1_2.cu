

#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void add(int *a, int *b, int *c) {
	// *c = *a + *b
	int id=threadIdx.x;
	c[id]=a[id]+b[id];

}
int main(void) {
	int a[2], b[2], c[2];
	// host copies of variables a, b & c
	int *d_a, *d_b, *d_c; // device copies of variables a, b & c
	int size = sizeof(int);
	// Allocate space for device copies of a, b, c
	hipMalloc((void **)&d_a, 2*size);
	hipMalloc((void **)&d_b, 2*size);
	hipMalloc((void **)&d_c, 2*size);
	// Setup input values
	a[0] = 3;
	a[1]=4;
	b[0] = 5;
	b[1]=5;
	// Copy inputs to device
	hipMemcpy(d_a, &a, 2*size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, 2*size, hipMemcpyHostToDevice);
	// Launch add() kernel on GPU

	add<<<1,2>>>(d_a, d_b, d_c);
	// Copy result back to host
	hipMemcpy(&c, d_c, 2*size, hipMemcpyDeviceToHost);

	printf("ANS	%d\t%d\n",c[0],c[1] );
	// Cleanup
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}